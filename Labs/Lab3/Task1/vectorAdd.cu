#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

// CUDA Kernel function for vector addition
__global__ void vectorAdd(const int *A, const int *B, int *C, int numElements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    int numElements = 1 << 20; // Increase the number of elements to 2^20
    size_t size = numElements * sizeof(int);

    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);
    int *h_C = (int *)malloc(size);

    for (int i = 0; i < numElements; ++i) {
        h_A[i] = i;
        h_B[i] = i;
    }

    int *d_A = NULL;
    hipMalloc((void **)&d_A, size);
    int *d_B = NULL;
    hipMalloc((void **)&d_B, size);
    int *d_C = NULL;
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Adjust the number of threads per block and blocks per grid
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording the time
    hipEventRecord(start, 0);

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Stop recording the time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < numElements; ++i) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    printf("Test PASSED\n");
    printf("Kernel execution time: %f ms\n", milliseconds);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}