#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// ================== Two Kernel Implementations ==================
// Grid-stride version
__global__ void vectorAdd_grid_stride(const int *A, const int *B, int *C, int numElements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = i; idx < numElements; idx += stride) {
        C[idx] = A[idx] + B[idx];
    }
}

// Traditional single-thread version
__global__ void vectorAdd_no_stride(const int *A, const int *B, int *C, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        C[idx] = A[idx] + B[idx];
    }
}

// ================== Performance Testing Framework ==================
int main() {
    const int numElements = 1 << 20;  // Fixed problem size
    size_t size = numElements * sizeof(int);
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    // Configure test parameters
    const int test_dims[] = {16, 32, 64, 128, 256};
    const int num_tests = sizeof(test_dims)/sizeof(test_dims[0]);

    // Initialize host memory
    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);
    int *h_C = (int *)malloc(size);
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = i;
        h_B[i] = i;
    }

    // Allocate device memory
    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Create CUDA event timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("| blk_dim | grid_dim (stride) | grid_dim (no-stride) | time_stride (ms) | time_no_stride (ms) |\n");
    printf("|---------|-------------------|----------------------|------------------|---------------------|\n");

    // Warm up
    vectorAdd_grid_stride<<<numSMs * 8 * 256 / 128, 128>>>(d_A, d_B, d_C, numElements);
    vectorAdd_no_stride<<<(numElements + 128 - 1) / 128, 128>>>(d_A, d_B, d_C, numElements);
    hipDeviceSynchronize();

    for (int i = 0; i < num_tests; ++i) {
        const int blk_dim = test_dims[i];
        
        // Calculate grid dimensions for both versions
        const int grid_stride = numSMs * 8 * 256 / blk_dim;
        const int grid_no_stride = (numElements + blk_dim - 1) / blk_dim;

        // Test grid-stride version
        float time_stride;
        hipMemset(d_C, 0, size);
        hipEventRecord(start);
        vectorAdd_grid_stride<<<grid_stride, blk_dim>>>(d_A, d_B, d_C, numElements);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_stride, start, stop);

        // Test no-stride version
        float time_no_stride;
        hipMemset(d_C, 0, size);
        hipEventRecord(start);
        vectorAdd_no_stride<<<grid_no_stride, blk_dim>>>(d_A, d_B, d_C, numElements);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_no_stride, start, stop);

        printf("| %7d | %17d | %20d | %16.3f | %19.3f |\n",
               blk_dim, grid_stride, grid_no_stride, time_stride, time_no_stride);
    }

    // Clean up resources
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}